
#include <hip/hip_runtime.h>
#include <stdio.h>

// Luis Miguel García Marín

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  //float *a;
  //float *b;
  float *c;
  // Punteros que apuntarán a posiciones de memoria con los que trabajará la CPU

  //a = (float *) malloc(size);
  //b = (float *) malloc(size);
  c = (float *) malloc(size);
  // Asignamos espacio en la memoria que trabaja la CPU

  float *da;
  float *db;
  float *dc;
  // Punteros que apuntan a posiciones de la memoria de vídeo con los que trabajará la GPU
  
  hipMalloc(&da, size);
  hipMalloc(&db, size);
  hipMalloc(&dc, size);
  // Asignamos espacio en la memoria de vídeo de la GPU

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  initWith<<<numberOfBlocks, threadsPerBlock>>>(3, da, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(4, db, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(0, dc, N);

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(dc, da, db, N);
  // Les pasamos los punteros de device (da,db,dc) y no de host (a,b,c)

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  //cudaMemcpy(a, da, size, cudaMemcpyDeviceToHost);
  //cudaMemcpy(b, db, size, cudaMemcpyDeviceToHost);
  hipMemcpy(c, dc, size, hipMemcpyDeviceToHost);
  // Copiamos los resultados (del último vector, el resultante) de la GPU a la CPU

  checkElementsAre(7, c, N);

  hipFree(da);
  hipFree(db);
  hipFree(dc);
  // Liberamos la memoria de vídeo
  
  //free(a);
  //free(b);
  free(c);
  // Liberamos la memoria principal que trabajaba la CPU
}
